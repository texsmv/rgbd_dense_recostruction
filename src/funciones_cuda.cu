#include "hip/hip_runtime.h"
#include "funciones_cuda.h"

__global__ void fillPoints(uint16_t* pSource, float* puntos, float depthFactor, float cx, float cy, float fx, float fy, int h, int w){

  int u = blockIdx.x * blockDim.x + threadIdx.x;
  int v = blockIdx.y * blockDim.y + threadIdx.y;

  if(u < w && v < h){
    int i = u + v * w;
    uint16_t value = pSource[i];
    if(value != 0){
      float pz = value / depthFactor;
      at_vec3(puntos, i, 2) = pz;
      at_vec3(puntos, i, 0) = (u - cx) * pz / fx;
      at_vec3(puntos, i, 1) = (v - cy) * pz / fy;
    }
    else{
      at_vec3(puntos, i, 2) = 0;
      at_vec3(puntos, i, 0) = 0;
      at_vec3(puntos, i, 1) = 0;
    }

  }
}
