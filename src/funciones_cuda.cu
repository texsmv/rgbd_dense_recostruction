#include "hip/hip_runtime.h"
#include "funciones_cuda.h"

__global__ void fillPoints(uint16_t* pSource, float* puntos, float depthFactor, float cx, float cy, float fx, float fy, int h, int w){

  int v = blockIdx.x * blockDim.x + threadIdx.x;
  int u = blockIdx.y * blockDim.y + threadIdx.y;

  if(u < w && v < h){
    int i = v + u * w;
    float value = pSource[i];
    if(value != 0){
      float pz = value / depthFactor;
      at_vec3(puntos, i, 2) = pz;
      at_vec3(puntos, i, 0) = (u - cx) * pz / fx;
      at_vec3(puntos, i, 1) = (v - cy) * pz / fy;
    }
    else{
      at_vec3(puntos, i, 2) = 0;
      at_vec3(puntos, i, 0) = 0;
      at_vec3(puntos, i, 1) = 0;
    }

  }
}
